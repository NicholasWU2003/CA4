#include "hip/hip_runtime.h"
/*
 * Skeleton code for use with Computer Architecture 2023 assignment 4,
 * LIACS, Leiden University.
 *
 * Task 3: Batched Processing. 
 *
 */

#include "image.h"
#include "experiment.hpp"

#include <iostream>

#include <cstring>
#include <cerrno>

#include <getopt.h>

#include <sys/types.h>
#include <dirent.h>

/* Some simple assert macro and inline function to handle CUDA-errors
 * more easily.
 */
#define CUDA_ASSERT(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void
cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
              file, line);

      if (abort)
        exit(code);
    }
}


/*
 * CUDA kernels
 *
 * "op_" functions below to be converted to CUDA kernels. Remember that
 * every kernel should be declared as `__global__ void`. Every function
 * you want to call from a kernel, should begin with `__device__`.
 *
 * op_cuda_copy is a CUDA kernel provided as example.
 */


/* Example kernel for an image copy operation. */
__global__ void
op_cuda_copy(uint32_t *dst, const uint32_t *src, const int rowstride,
             const int x, const int y,
             const int width, const int height)
{
  const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx < x || xx >= width || yy < y || y >= height)
    return;

  /* Get the pixel in src and store in dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, xx, yy) = pixel;
}


/*
 * CPU kernels
 *
 * Leave these functions in place, they are required for the "test
 * mode" to work. Make a copy in order to convert to a CUDA kernel.
 *
 */


/* Tiles the @tile image on @background using alpha blending. For the tile
 * an alpha value of @tile_alpha is used.
 */
/* Do not remove this function, it is required for the "test mode" to work. */
void
op_tile_composite(image_t *background,
                  const image_t *tile, const float tile_alpha)
{
  for (int y = 0; y < background->height; y++)
    {
      for (int x = 0; x < background->width; x++)
        {
          int tx = x % tile->width;
          int ty = y % tile->height;

          rgba_t dst, src;
          RGBA_unpack(dst, *image_get_pixel(background, x, y));
          RGBA_unpack(src, *image_get_pixel(tile, tx, ty));
          RGBA_mults(src, src, tile_alpha);
          RGBA_mults(dst, dst, 1.f - tile_alpha);
          RGBA_add(dst, dst, src);
          RGBA_pack(*image_get_pixel(background, x, y), dst);
        }
    }
}



// Batched global memory kernel
__global__ void tile_global_batched(uint32_t **dst_batch, int bg_width, int bg_height, int bg_rowbytes,
                                    const uint32_t *tile, int tile_width, int tile_height, int tile_rowbytes,
                                    const float alpha, int batch_size)
{
    // Compute the (x, y) coordinates of the pixel this thread will process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Exit if the thread is outside the bounds of the background image
    if (x >= bg_width || y >= bg_height) return;

    // Calculate the number of uint32_t elements per row for background and tile
    const int bg_pitch   = bg_rowbytes   >> 2;      // rowstride in uint32_t words
    const int tile_pitch = tile_rowbytes >> 2;

    // Compute the corresponding tile coordinates (tx, ty) using modulo for tiling
    int tx = x % tile_width;
    int ty = y % tile_height;
    // Fetch the tile pixel at (tx, ty)
    uint32_t tile_pix = tile[ty * tile_pitch + tx];

    // Unpack tile pixel once
    rgba_t t;
    RGBA_unpack(t, tile_pix);
    RGBA_mults(t, t, alpha);  // Scale tile by alpha once

    // Process all images in the batch
    for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
        uint32_t *dst = dst_batch[batch_idx];
        
        // Fetch the background pixel at (x, y)
        uint32_t bg_pix = dst[y * bg_pitch + x];

        // Unpack background pixel
        rgba_t b;
        RGBA_unpack(b, bg_pix);

        // Perform alpha blending
        rgba_t out;
        RGBA_mults(b, b, 1.f - alpha);   // Scale background by (1 - alpha)
        RGBA_add(out, b, t);             // Add the two results

        // Pack the blended result back into uint32_t and store it
        RGBA_pack(bg_pix, out);
        dst[y * bg_pitch + x] = bg_pix;
    }
}

// Batched shared memory kernel
__global__ void tile_shared_batched(uint32_t **dst_batch, int bg_width, int bg_height, int bg_rowbytes,
                                    const uint32_t *tile, int tile_width, int tile_height, int tile_rowbytes,
                                    const float alpha, int batch_size)
{
    // Shared memory for the tile (allocated dynamically)
    extern __shared__ uint32_t tile_shmem[];

    // Calculate the number of uint32_t elements per row for tile
    const int tile_pitch = tile_rowbytes >> 2;

    // Each thread loads one or more tile pixels into shared memory
    int tile_size = tile_width * tile_height;
    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    int block_threads = blockDim.x * blockDim.y;
    for (int i = thread_id; i < tile_size; i += block_threads) {
        int tx = i % tile_width;
        int ty = i / tile_width;
        tile_shmem[i] = tile[ty * tile_pitch + tx];
    }
    __syncthreads();

    // Compute the (x, y) coordinates of the pixel this thread will process
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= bg_width || y >= bg_height) return;

    // Calculate the number of uint32_t elements per row for background
    const int bg_pitch = bg_rowbytes >> 2;

    // Compute the corresponding tile coordinates (tx, ty) using modulo for tiling
    int tx = x % tile_width;
    int ty = y % tile_height;
    // Fetch the tile pixel from shared memory
    uint32_t tile_pix = tile_shmem[ty * tile_width + tx];

    // Unpack tile pixel once
    rgba_t t;
    RGBA_unpack(t, tile_pix);
    RGBA_mults(t, t, alpha);  // Scale tile by alpha once

    // Process all images in the batch
    for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
        uint32_t *dst = dst_batch[batch_idx];
        
        // Fetch the background pixel at (x, y)
        uint32_t bg_pix = dst[y * bg_pitch + x];

        // Unpack background pixel
        rgba_t b;
        RGBA_unpack(b, bg_pix);

        // Perform alpha blending
        rgba_t out;
        RGBA_mults(b, b, 1.f - alpha);   // Scale background by (1 - alpha)
        RGBA_add(out, b, t);             // Add the two results

        // Pack the blended result back into uint32_t and store it
        RGBA_pack(bg_pix, out);
        dst[y * bg_pitch + x] = bg_pix;
    }
}

/* Returns elapsed time in msec */
static float
run_cuda_kernels(image_t *background[], const size_t nImages,
                 const image_t *tile)
{
  /* Allocate device memory for background images and tile */
  uint32_t *d_tile = nullptr;
  size_t bg_bytes = background[0]->rowstride * background[0]->height;
  size_t tile_bytes = tile->rowstride * tile->height;

  // Allocate memory for tile
  hipMalloc(&d_tile, tile_bytes);
  hipMemcpy(d_tile, tile->data, tile_bytes, hipMemcpyHostToDevice);

  // For batched processing: allocate array of device pointers
  uint32_t **d_bg_array = nullptr;
  uint32_t **h_bg_pointers = new uint32_t*[nImages];
  
  // Allocate device memory for each background image
  for (size_t i = 0; i < nImages; i++) {
    hipMalloc(&h_bg_pointers[i], bg_bytes);
    hipMemcpy(h_bg_pointers[i], background[i]->data, bg_bytes, hipMemcpyHostToDevice);
  }
  
  // Allocate device memory for the array of pointers
  hipMalloc(&d_bg_array, nImages * sizeof(uint32_t*));
  hipMemcpy(d_bg_array, h_bg_pointers, nImages * sizeof(uint32_t*), hipMemcpyHostToDevice);

  /* Calculate block size and grid dimensions */
  dim3 block(16, 16); 
  int PPTx = 1;
  int PPTy = 1;
  size_t sharedMem = tile->width * tile->height * sizeof(uint32_t);

  dim3 grid((background[0]->width + (block.x * PPTx - 1)) / (block.x * PPTx),
            (background[0]->height + (block.y * PPTy - 1)) / (block.y * PPTy));

  /* Timing setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  
    // For batch processing, use batched kernels
    // tile_global_batched<<<grid, block>>>(d_bg_array, background[0]->width, background[0]->height, background[0]->rowstride,
    //                                       d_tile, tile->width, tile->height, tile->rowstride, 0.2f, nImages);
    
    // batched shared memory kernel
    tile_shared_batched<<<grid, block, sharedMem>>>(d_bg_array, background[0]->width, background[0]->height, background[0]->rowstride,
                                                     d_tile, tile->width, tile->height, tile->rowstride, 0.2f, nImages);
  

  CUDA_ASSERT( hipGetLastError() );

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));

  /* Copy results back */
  for (size_t i = 0; i < nImages; i++) {
    hipMemcpy(background[i]->data, h_bg_pointers[i], bg_bytes, hipMemcpyDeviceToHost);
    hipFree(h_bg_pointers[i]);
  }

  /* Cleanup */
  hipFree(d_bg_array);
  hipFree(d_tile);
  delete[] h_bg_pointers;

  return msec;
}

static void
run_test(const std::string &infilename, image_t *tile,
         const std::string &outfilename)
{
  std::cout << "Testing with " << infilename << " ...\n";

  image_t *background[1];
  background[0] = image_new_from_pngfile(infilename.c_str());
  if (!background[0])
    return;

  /* Create a copy to be manipulated on CPU */
  image_t *original = image_new_from_image(background[0]);
  std::memcpy(original->data, background[0]->data,
              background[0]->rowstride * background[0]->height);

  /* Run CPU kernels */
  op_tile_composite(original, tile, 0.2f);

  /* Run GPU kernels */
  run_cuda_kernels(background, 1, tile);

  /* Compare the results */
  const int max_error = 64;
  int errors = image_compare(background[0], original, max_error);
  if (errors > 0)
    std::cerr << "Images do not match, " << errors
              << " errors detected (max=" << max_error << ").\n";
  else
    std::cerr << "Images match.\n";

  /* Write GPU result to PNG if requested */
  if (not outfilename.empty())
    {
      image_save_as_pngfile(background[0], outfilename.c_str());
      std::cerr << "Wrote GPU result to " << outfilename << "\n";
    }

  image_free(original);
  image_free(background[0]);
}

/* Process a single image, or set/batch of images.. Warning: does not detect
 * errors.
 */
static void
process_images(size_t i, ExperimentTimer &timer,
               const std::vector<std::string> &infilenames,
               const image_t *tile,
               const std::vector<std::string> &outfilenames)
{
  /* Load image */
  image_t *background[infilenames.size()];

  auto startTime = ExperimentTimer::now();
  for (size_t j = 0; j < infilenames.size(); ++j)
    {
      background[j] = image_new_from_pngfile(infilenames[j].c_str());
      if (!background[j])
        return;
    }
  auto endTime = ExperimentTimer::now();

  timer.setLoadTime(i, endTime, startTime);

  float msec = run_cuda_kernels(background, infilenames.size(), tile);
  timer.setComputeTime(i, msec / 1000.);

  /* Save results if desired and if applicable */
  if (not outfilenames.empty())
    {
      for (size_t j = 0; j < outfilenames.size(); ++j)
        image_save_as_pngfile(background[j], outfilenames[j].c_str());
    }

  for (size_t j = 0; j < infilenames.size(); ++j)
    image_free(background[j]);
}

/* Code to run a single experiment, depending on batchSize parameter. */
static void
run_experiment(Experiment &exp,
               const std::string &indir, const std::string &outdir,
               image_t *tile, bool silentMode)
{
  ExperimentTimer timer(exp.addMeasurement());

  timer.start();

  for (size_t i = 0; i < exp.getNBatches(); ++i)
    {
      size_t count = exp.getBatchSize(i);

      std::vector<std::string> infilenames;
      std::vector<std::string> outfilenames;

      for (size_t j = 0; j < count; ++j)
        {
          infilenames.emplace_back(indir + std::string("/") + exp.getFrameFile(i, j));
          if (not outdir.empty())
            outfilenames.emplace_back(outdir + std::string("/") + exp.getFrameFile(i, j));
        }

      // if (not silentMode)
      //   {
      //     if (count == 1)
      //       std::cout << "Processing " << infilenames[0] << " ...\n" << std::flush;
      //     else
      //       std::cout << "Processing " << infilenames[0] << " - "
      //                 << infilenames[count - 1] << "...\n" << std::flush;
      //   }

      process_images(i, timer, infilenames, tile, outfilenames);
    }

  /* Note that the full timing of the experiment will include image
   * loading & saving time and memory transfers to and from the GPU.
   * The memory transfers are not counted in the runtime of the
   * individual images.
   */
  timer.end();

  /* Print statistics */
  if (not silentMode)
    {
      std::cout << "====\n";
      exp.printCPUUtilization(exp.back(), std::cout);
      exp.printStatistics(exp.back(), std::cout);
      std::cout << "====\n";
    }
}

/*
 * Main function
 */

static bool
has_png_extension(const std::string &filename)
{
  if (filename.size() < 4)
    return false;

  return filename.substr(filename.size() - 4, 4) == ".png";
}

static void
show_help(const char *progName)
{
  std::cerr << "usage: " << progName << " [-t] [-s] [-c] [-r REPEAT] [-b BATCH_SIZE] [-1] <indir> <tilefile> [outdir]\n"
            << "\n  where <indir>, and [outdir] are directories containing PNG files."
            << "\n  <tilefile> is a PNG file."
            << "\n  [outdir] is an optional parameter.\n"
            << "\n  -t  test mode: only processes the first image found in <indir> and compares"
            << "\n      the GPU result to the CPU result. Optionally outputs the GPU result if"
            << "\n      outdir is specified."
            << "\n  -s  disables output of experiment summaries (silent mode)."
            << "\n  -c  outputs an overview of all experiment results in CSV format."
            << "\n  -r  configures the number of times the experiment is repeated."
            << "\n  -b  configures the batch size (defaults to 1)."
            << "\n  -1  stops the experiment after processing one batch (defaults to false).\n";
}

int
main(int argc, char **argv)
{
  char c;
  long int n_repeat = 1;
  long int batchSize = 1;
  bool singleBatch = false;
  bool csvOutput = false;
  bool silentMode = false;
  bool testMode = false;

  /* Command line parsing */
  const char *progName = argv[0];

  while ((c = getopt(argc, argv, "r:b:1cst")) != -1)
    {
      switch (c)
        {
          case 'r':
            n_repeat = std::strtol(optarg, NULL, 10);
            if (errno > 0 || n_repeat == 0)
              {
                std::cerr << "Could not convert n_repeat argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case 'b':
            batchSize = std::strtol(optarg, NULL, 10);
            if (errno > 0 || batchSize == 0)
              {
                std::cerr << "Could not convert batchSize argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case '1':
            singleBatch = true;
            break;

          case 'c':
            csvOutput = true;
            break;

          case 's':
            silentMode = true;
            break;

          case 't':
            testMode = true;
            break;

          case 'h':
          default:
            show_help(progName);
            return EXIT_FAILURE;
            break;
        }
    }

  argc -= optind;
  argv += optind;

  if (argc < 2)
    {
      show_help(progName);
      return EXIT_FAILURE;
    }

  const char *indir = argv[0];
  const char *tilefile = argv[1];
  const char *outdir = argv[2];

  /* Check and open directories */
  DIR *indirp = opendir(indir);
  if (indirp == NULL)
    {
      const char *err = strerror(errno);
      std::cerr << "error: could not open directory '" << indir
                << "': " << err << "\n";
      return EXIT_FAILURE;
    }

  DIR *outdirp = NULL;
  if (outdir)
    {
      outdirp = opendir(outdir);
      if (outdirp == NULL)
        {
          const char *err = strerror(errno);
          std::cerr << "error: could not open directory '" << outdir
              << "': " << err << "\n";
          closedir(indirp);
          return EXIT_FAILURE;
        }

      closedir(outdirp);
    }

  /* Createa a list pf PNG files in given directory. */
  std::vector<std::string> framefiles;
  for (struct dirent *ent = readdir(indirp);
       ent != NULL; ent = readdir(indirp))
    {
      std::string filename(ent->d_name);
      if (has_png_extension(filename))
        framefiles.emplace_back(std::move(filename));
    }

  closedir(indirp);

  /* Load tile image */
  image_t *tile = image_new_from_pngfile(tilefile);
  if (!tile)
    return EXIT_FAILURE;

  if (testMode)
    {
      /* Test mode: process the first image found in "indir" and
       * compare the GPU result to the CPU result.
       */
      std::string infilename = std::string(indir) + std::string("/") + framefiles[0];
      std::string outfilename = outdir ? std::string(outdir) + std::string("/") + framefiles[0] : std::string();

      run_test(infilename, tile, outfilename);
    }
  else
    {
      /* Run experiment the requested number of times. */
      Experiment exp(framefiles, batchSize, singleBatch);
      for (int i = 0; i < n_repeat; ++i)
        run_experiment(exp, std::string(indir),
                       outdir ? std::string(outdir) : std::string(),
                       tile, silentMode);

      if (csvOutput)
        exp.outputCSVData(std::cout);
    }

  /* Clean up */
  image_free(tile);

  return EXIT_SUCCESS;
} 