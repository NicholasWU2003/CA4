#include "hip/hip_runtime.h"
/*
 * Skeleton code for use with Computer Architecture 2023 assignment 4,
 * LIACS, Leiden University.
 *
 * Task 1: grayscale kernel
 *
 */

#include "image.h"
#include "experiment.hpp"

#include <iostream>

#include <cstring>
#include <cerrno>

#include <getopt.h>

#include <sys/types.h>
#include <dirent.h>

/* Some simple assert macro and inline function to handle CUDA-errors
 * more easily.
 */
#define CUDA_ASSERT(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void
cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
              file, line);

      if (abort)
        exit(code);
    }
}


/*
 * CUDA kernels
 *
 * "op_" functions below to be converted to CUDA kernels. Remember that
 * every kernel should be declared as `__global__ void`. Every function
 * you want to call from a kernel, should begin with `__device__`.
 *
 * op_cuda_copy is a CUDA kernel provided as example.
 */


/* Example kernel for an image copy operation. */
__global__ void
op_cuda_copy(uint32_t *dst, const uint32_t *src, const int rowstride,
             const int x, const int y,
             const int width, const int height)
{
  const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx < x || xx >= width || yy < y || y >= height)
    return;

  /* Get the pixel in src and store in dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, xx, yy) = pixel;
}


/*
 * CPU kernels
 *
 * Leave these functions in place, they are required for the "test
 * mode" to work. Make a copy in order to convert to a CUDA kernel.
 *
 */


/* Computes the intensity of @color. To do so, we use CIE 1931 weights
 * multiplied by alpha: Y = A( 0.2126R + 0.7152G + 0.0722B ).
 */
static inline float
compute_intensity(rgba_t color)
{
  return color.w * (0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z);
}

/* Computes the grayscale value for each pixel in @src and stores this in @dst.
 * @dst is expected to have been created already with the correct dimensions.
 * Safe to use a in-place operation.
 */
/* Do not remove this function, it is required for the "test mode" to work. */
void
op_grayscale(image_t *dst, const image_t *src)
{
  for (int x = 0; x < dst->width; x++)
    {
      for (int y = 0; y < dst->height; y++)
        {
          rgba_t color, gray;
          RGBA_unpack(color, *image_get_pixel(src, x, y));
          float intensity = compute_intensity(color);
          RGBA(gray, intensity, intensity, intensity, 1.f);
          RGBA_pack(*image_get_pixel(dst, x, y), gray);
        }
    }
}

__global__ void
op_cuda_grayscale_rq1(uint32_t *dst, const uint32_t *src, int rowstride,
                      int width, int height)
{
    // Calculate the thread's unique position in the grid
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    printf("Thread (%d, %d) accessing pixel\n", x, y);

    // Ensure thread is within image bounds
    if (x >= width || y >= height) return;

    // Fetch the color pixel
    rgba_t color, gray;
    RGBA_unpack(color, *image_get_pixel_data(src, rowstride, x, y));

    // Compute intensity
    float intensity = color.w * (0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z);
    RGBA(gray, intensity, intensity, intensity, 1.f);

    // Store the grayscale pixel
    RGBA_pack(*image_get_pixel_data(dst, rowstride, x, y), gray);
}






// /* Returns elapsed time in msec */
// static float
// run_cuda_kernel(image_t *background)
// {

//   if (!background || !background->data) {
//         std::cerr << "Error: Invalid background image.\n";
//         return -1;
//   }

//   /* TODO: allocate buffers to contain background image. */
//   uint32_t *input, *output;
//   int rowstride = background->rowstride;
//   int width = background->width;
//   int height = background->height;

//   if (rowstride <= 0 || height <= 0) {
//     std::cerr << "Error: Invalid rowstride or height.\n";
//     return -1;
//   } 

//   size_t imageSize = rowstride * width * sizeof(uint32_t); 
//   CUDA_ASSERT(hipMalloc(&input, imageSize));
//   CUDA_ASSERT(hipMalloc(&output, imageSize));


//   /* TODO: copy the input image to the background buffer allocated
//    * on the GPU.
//    */
//    CUDA_ASSERT(hipMemcpy(input, background->data, imageSize, hipMemcpyHostToDevice));


//   /* TODO: calculate the block size and number of thread blocks. */
//   const dim3 blockSize(8,8); //8*8=64 threads per block
//   const dim3 gridSize((background->width + blockSize.x - 1) / blockSize.x,(background->height + blockSize.y - 1) / blockSize.y);


//   /* "computetime" will only include the actual time taken by the GPU
//    * to perform the image operation. So, this excludes image loading,
//    * saving and the memory transfers to/from the GPU.
//    */
//   hipEvent_t start, stop;
//   hipEventCreate(&start);
//   hipEventCreate(&stop);

//   /* Start the timer */
//   CUDA_ASSERT(hipEventRecord(start));

//   /* TODO: replace with CUDA kernel call. If you have multiple variants
//    * of the kernel, you can choose which one to run here. Or make copies
//    * of this run_cuda_kernel() function.
//    */
// // #if 0
// //   op_grayscale(background, background); /* in-place */
// // #endif

//   //launch CUDA kernel
//   op_cuda_grayscale_rq1<<<gridSize, blockSize>>>(output, input, background->rowstride, background->width, background->height);

//   CUDA_ASSERT( hipGetLastError() );

//   /* Stop timer */
//   CUDA_ASSERT(hipEventRecord(stop));
//   CUDA_ASSERT(hipEventSynchronize(stop));

//   float msec = 0;
//   CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));

//   /* TODO: copy the result buffer back to CPU host memory. */
//   CUDA_ASSERT(hipMemcpy(background->data, output, imageSize, hipMemcpyDeviceToHost));

//   /* TODO: release GPU memory */
//   CUDA_ASSERT(hipFree(input));
//   CUDA_ASSERT(hipFree(output));

//   return msec;
// }

static float
run_cuda_kernel(image_t *background)
{
    if (!background || !background->data) {
        std::cerr << "Error: Invalid background image or data.\n";
        return -1;
    }

    int rowstride = background->rowstride;
    int width = background->width;
    int height = background->height;

    if (rowstride <= 0 || height <= 0 || width <= 0) {
        std::cerr << "Error: Invalid image dimensions.\n";
        return -1;
    }

    // Debug: Log image properties
    std::cerr << "Image Properties: width=" << width
              << ", height=" << height
              << ", rowstride=" << rowstride << "\n";

    // Allocate GPU memory
    uint32_t *input, *output;
    size_t imageSize = rowstride * height * sizeof(uint32_t);

    hipError_t err = hipMalloc(&input, imageSize);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc for input failed: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    err = hipMalloc(&output, imageSize);
    if (err != hipSuccess) {
        std::cerr << "hipMalloc for output failed: " << hipGetErrorString(err) << "\n";
        hipFree(input); // Cleanup
        return -1;
    }

    // Copy input image to GPU
    std::cerr << "Copying input data to device...\n";
    err = hipMemcpy(input, background->data, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy HostToDevice failed: " << hipGetErrorString(err) << "\n";
        hipFree(input);
        hipFree(output);
        return -1;
    }
    std::cerr << "Input data copied successfully.\n";

    // Configure kernel launch
    const dim3 blockSize(8, 8);
    const dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                        (height + blockSize.y - 1) / blockSize.y);

    std::cerr << "Grid Size: (" << gridSize.x << ", " << gridSize.y << ")\n";
    std::cerr << "Block Size: (" << blockSize.x << ", " << blockSize.y << ")\n";

    // Launch kernel
    hipEvent_t start, stop;
    CUDA_ASSERT(hipEventCreate(&start));
    CUDA_ASSERT(hipEventCreate(&stop));
    CUDA_ASSERT(hipEventRecord(start));

    op_cuda_grayscale_rq1<<<gridSize, blockSize>>>(output, input, rowstride, width, height);
    err = hipGetLastError(); // Check for launch errors
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << "\n";
        hipFree(input);
        hipFree(output);
        return -1;
    }

    hipDeviceSynchronize(); // Ensure kernel execution completes

    CUDA_ASSERT(hipEventRecord(stop));
    CUDA_ASSERT(hipEventSynchronize(stop));

    // Measure elapsed time
    float msec = 0;
    CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));

    std::cerr << "Kernel execution completed in " << msec << " ms.\n";

    // Copy output data back to host
    std::cerr << "Copying output data back to host...\n";
    err = hipMemcpy(background->data, output, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "hipMemcpy DeviceToHost failed: " << hipGetErrorString(err) << "\n";
        hipFree(input);
        hipFree(output);
        return -1;
    }
    std::cerr << "Output data copied successfully.\n";

    // // Free GPU memory
    // std::cerr << "Freeing GPU memory...\n";
    // if (input) {
    //   std::cerr << "Freeing input...\n";
    //     err = hipFree(input);
    //     std::cerr << "Freeing input 2 ...\n";
    //     if (err != hipSuccess) {
    //         std::cerr << "hipFree for input failed: " << hipGetErrorString(err) << "\n";
    //     } else {
    //         std::cerr << "Freeing input completed.\n";
    //     }
    // }

    // if (output) {
    //     err = hipFree(output);
    //     if (err != hipSuccess) {
    //         std::cerr << "hipFree for output failed: " << hipGetErrorString(err) << "\n";
    //     } else {
    //         std::cerr << "Freeing output completed.\n";
    //     }
    // }

    // std::cerr << "GPU memory freed.\n";
    return msec;
}








static void
run_test(const std::string &infilename,
         const std::string &outfilename)
{
  std::cout << "Testing with " << infilename << " ...\n";

  image_t *background;
  background = image_new_from_pngfile(infilename.c_str());
  if (!background)
    std::cerr << "Could not load image " << infilename << ".\n";
    return;

  /* Create a copy to be manipulated on CPU */
  image_t *original = image_new_from_image(background);
  std::memcpy(original->data, background->data,
              background->rowstride * background->height);

  /* Run CPU kernels */
  op_grayscale(original, original); /* in-place */

  /* Run GPU kernels */
  run_cuda_kernel(background);

  /* Compare the results */
  const int max_error = 64;
  int errors = image_compare(background, original, max_error);
  if (errors > 0)
    std::cerr << "Images do not match, " << errors
              << " errors detected (max=" << max_error << ").\n";
  else
    std::cerr << "Images match.\n";

  /* Write GPU result to PNG if requested */
  if (not outfilename.empty())
    {
      image_save_as_pngfile(background, outfilename.c_str());
      std::cerr << "Wrote GPU result to " << outfilename << "\n";
    }

  image_free(original);
  image_free(background);
}


/* Process a single image. Warning: does not detect errors.
 */
static void
process_images(size_t i, ExperimentTimer &timer,
               const std::string &infilename,
               const std::string &outfilename)
{
  /* Load image */
  image_t *background;

  auto startTime = ExperimentTimer::now();
  background = image_new_from_pngfile(infilename.c_str());
  if (!background)
    return;
  auto endTime = ExperimentTimer::now();

  timer.setLoadTime(i, endTime, startTime);

  float msec = run_cuda_kernel(background);
  timer.setComputeTime(i, msec / 1000.);

  /* Save results if desired and if applicable */
  if (not outfilename.empty())
    image_save_as_pngfile(background, outfilename.c_str());

  image_free(background);
}


/* Code to run a single experiment, depending on batchSize parameter. */
static void
run_experiment(Experiment &exp,
               const std::string &infile, const std::string &outfile,
               bool silentMode)
{
  ExperimentTimer timer(exp.addMeasurement());

  timer.start();

  if (not silentMode)
    std::cout << "Processing " << infile << " ...\n" << std::flush;

  process_images(0, timer, infile, outfile);

  /* Note that the full timing of the experiment will include image
   * loading & saving time and memory transfers to and from the GPU.
   * The memory transfers are not counted in the runtime of the
   * individual images.
   */
  timer.end();

  /* Print statistics */
  if (not silentMode)
    {
      std::cout << "====\n";
      exp.printCPUUtilization(exp.back(), std::cout);
      exp.printStatistics(exp.back(), std::cout);
      std::cout << "====\n";
    }
}


/*
 * Main function
 */

static void
show_help(const char *progName)
{
  std::cerr << "usage: " << progName << " [-t] [-s] [-c] [-r REPEAT] <infile> [outfile]\n"
            << "\n  where <infile>, and [outfile] are PNG files."
            << "\n  [outfile] is an optional parameter.\n"
            << "\n  -t  test mode: compares the GPU result to the CPU result. Optionally outputs "
            << "\n      the GPU results if outfile is specified."
            << "\n  -s  disables output of experiment summaries (silent mode)."
            << "\n  -c  outputs an overview of all experiment results in CSV format."
            << "\n  -r  configures the number of times the experiment is repeated.\n";
}


int
main(int argc, char **argv)
{
  char c;
  long int n_repeat = 1;
  bool csvOutput = false;
  bool silentMode = false;
  bool testMode = false;

  /* Command line parsing */
  const char *progName = argv[0];

  while ((c = getopt(argc, argv, "r:b:1cst")) != -1)
    {
      switch (c)
        {
          case 'r':
            n_repeat = std::strtol(optarg, NULL, 10);
            if (errno > 0 || n_repeat == 0)
              {
                std::cerr << "Could not convert n_repeat argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case 'c':
            csvOutput = true;
            break;

          case 's':
            silentMode = true;
            break;

          case 't':
            testMode = true;
            break;

          case 'h':
          default:
            show_help(progName);
            return EXIT_FAILURE;
            break;
        }
    }

  argc -= optind;
  argv += optind;

  if (argc < 1)
    {
      show_help(progName);
      return EXIT_FAILURE;
    }

  const char *infile = argv[0];
  const char *outfile = argv[1];

  if (testMode)
    {
      /* Test mode: process the first image found in "indir" and
       * compare the GPU result to the CPU result.
       */
      std::string infilename(infile);
      std::string outfilename = outfile ? std::string(outfile) : std::string();

      run_test(infilename, outfilename);
    }
  else
    {
      /* Run experiment the requested number of times. */
      std::vector<std::string> files {infile};
      Experiment exp(files, 1, true);
      for (int i = 0; i < n_repeat; ++i)
        run_experiment(exp, std::string(infile),
                       outfile ? std::string(outfile) : std::string(),
                       silentMode);

      if (csvOutput)
        exp.outputCSVData(std::cout);
    }

  return EXIT_SUCCESS;
}
