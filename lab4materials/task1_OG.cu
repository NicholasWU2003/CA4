#include "hip/hip_runtime.h"
/*
 * Skeleton code for use with Computer Architecture 2023 assignment 4,
 * LIACS, Leiden University.
 *
 * Task 1: grayscale kernel
 *
 */

#include "image.h"
#include "experiment.hpp"

#include <iostream>

#include <cstring>
#include <cerrno>

#include <getopt.h>

#include <sys/types.h>
#include <dirent.h>

/* Some simple assert macro and inline function to handle CUDA-errors
 * more easily.
 */
#define CUDA_ASSERT(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void
cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
              file, line);

      if (abort)
        exit(code);
    }
}


/*
 * CUDA kernels
 *
 * "op_" functions below to be converted to CUDA kernels. Remember that
 * every kernel should be declared as `__global__ void`. Every function
 * you want to call from a kernel, should begin with `__device__`.
 *
 * op_cuda_copy is a CUDA kernel provided as example.
 */


/* Example kernel for an image copy operation. */
__global__ void
op_cuda_copy(uint32_t *dst, const uint32_t *src, const int rowstride,
             const int x, const int y,
             const int width, const int height)
{
  const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx < x || xx >= width || yy < y || y >= height)
    return;

  /* Get the pixel in src and store in dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, xx, yy) = pixel;
}


/*
 * CPU kernels
 *
 * Leave these functions in place, they are required for the "test
 * mode" to work. Make a copy in order to convert to a CUDA kernel.
 *
 */


/* Computes the intensity of @color. To do so, we use CIE 1931 weights
 * multiplied by alpha: Y = A( 0.2126R + 0.7152G + 0.0722B ).
 */
static inline float
compute_intensity(rgba_t color)
{
  return color.w * (0.2126f * color.x + 0.7152f * color.y + 0.0722f * color.z);
}

/* Computes the grayscale value for each pixel in @src and stores this in @dst.
 * @dst is expected to have been created already with the correct dimensions.
 * Safe to use a in-place operation.
 */
/* Do not remove this function, it is required for the "test mode" to work. */
void
op_grayscale(image_t *dst, const image_t *src)
{
  for (int x = 0; x < dst->width; x++)
    {
      for (int y = 0; y < dst->height; y++)
        {
          rgba_t color, gray;
          RGBA_unpack(color, *image_get_pixel(src, x, y));
          float intensity = compute_intensity(color);
          RGBA(gray, intensity, intensity, intensity, 1.f);
          RGBA_pack(*image_get_pixel(dst, x, y), gray);
        }
    }
}


/* Returns elapsed time in msec */
static float
run_cuda_kernel(image_t *background)
{
  /* TODO: allocate buffers to contain background image. */
  

  /* TODO: copy the input image to the background buffer allocated
   * on the GPU.
   */

  /* TODO: calculate the block size and number of thread blocks. */


  /* "computetime" will only include the actual time taken by the GPU
   * to perform the image operation. So, this excludes image loading,
   * saving and the memory transfers to/from the GPU.
   */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Start the timer */
  CUDA_ASSERT(hipEventRecord(start));

  /* TODO: replace with CUDA kernel call. If you have multiple variants
   * of the kernel, you can choose which one to run here. Or make copies
   * of this run_cuda_kernel() function.
   */
#if 0
  op_grayscale(background, background); /* in-place */
#endif


  CUDA_ASSERT( hipGetLastError() );

  /* Stop timer */
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));

  /* TODO: copy the result buffer back to CPU host memory. */

  /* TODO: release GPU memory */

  return msec;
}


static void
run_test(const std::string &infilename,
         const std::string &outfilename)
{
  std::cout << "Testing with " << infilename << " ...\n";

  image_t *background;
  background = image_new_from_pngfile(infilename.c_str());
  if (!background)
    return;

  /* Create a copy to be manipulated on CPU */
  image_t *original = image_new_from_image(background);
  std::memcpy(original->data, background->data,
              background->rowstride * background->height);

  /* Run CPU kernels */
  op_grayscale(original, original); /* in-place */

  /* Run GPU kernels */
  run_cuda_kernel(background);

  /* Compare the results */
  const int max_error = 64;
  int errors = image_compare(background, original, max_error);
  if (errors > 0)
    std::cerr << "Images do not match, " << errors
              << " errors detected (max=" << max_error << ").\n";
  else
    std::cerr << "Images match.\n";

  /* Write GPU result to PNG if requested */
  if (not outfilename.empty())
    {
      image_save_as_pngfile(background, outfilename.c_str());
      std::cerr << "Wrote GPU result to " << outfilename << "\n";
    }

  image_free(original);
  image_free(background);
}


/* Process a single image. Warning: does not detect errors.
 */
static void
process_images(size_t i, ExperimentTimer &timer,
               const std::string &infilename,
               const std::string &outfilename)
{
  /* Load image */
  image_t *background;

  auto startTime = ExperimentTimer::now();
  background = image_new_from_pngfile(infilename.c_str());
  if (!background)
    return;
  auto endTime = ExperimentTimer::now();

  timer.setLoadTime(i, endTime, startTime);

  float msec = run_cuda_kernel(background);
  timer.setComputeTime(i, msec / 1000.);

  /* Save results if desired and if applicable */
  if (not outfilename.empty())
    image_save_as_pngfile(background, outfilename.c_str());

  image_free(background);
}


/* Code to run a single experiment, depending on batchSize parameter. */
static void
run_experiment(Experiment &exp,
               const std::string &infile, const std::string &outfile,
               bool silentMode)
{
  ExperimentTimer timer(exp.addMeasurement());

  timer.start();

  if (not silentMode)
    std::cout << "Processing " << infile << " ...\n" << std::flush;

  process_images(0, timer, infile, outfile);

  /* Note that the full timing of the experiment will include image
   * loading & saving time and memory transfers to and from the GPU.
   * The memory transfers are not counted in the runtime of the
   * individual images.
   */
  timer.end();

  /* Print statistics */
  if (not silentMode)
    {
      std::cout << "====\n";
      exp.printCPUUtilization(exp.back(), std::cout);
      exp.printStatistics(exp.back(), std::cout);
      std::cout << "====\n";
    }
}


/*
 * Main function
 */

static void
show_help(const char *progName)
{
  std::cerr << "usage: " << progName << " [-t] [-s] [-c] [-r REPEAT] <infile> [outfile]\n"
            << "\n  where <infile>, and [outfile] are PNG files."
            << "\n  [outfile] is an optional parameter.\n"
            << "\n  -t  test mode: compares the GPU result to the CPU result. Optionally outputs "
            << "\n      the GPU results if outfile is specified."
            << "\n  -s  disables output of experiment summaries (silent mode)."
            << "\n  -c  outputs an overview of all experiment results in CSV format."
            << "\n  -r  configures the number of times the experiment is repeated.\n";
}


int
main(int argc, char **argv)
{
  char c;
  long int n_repeat = 1;
  bool csvOutput = false;
  bool silentMode = false;
  bool testMode = false;

  /* Command line parsing */
  const char *progName = argv[0];

  while ((c = getopt(argc, argv, "r:b:1cst")) != -1)
    {
      switch (c)
        {
          case 'r':
            n_repeat = std::strtol(optarg, NULL, 10);
            if (errno > 0 || n_repeat == 0)
              {
                std::cerr << "Could not convert n_repeat argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case 'c':
            csvOutput = true;
            break;

          case 's':
            silentMode = true;
            break;

          case 't':
            testMode = true;
            break;

          case 'h':
          default:
            show_help(progName);
            return EXIT_FAILURE;
            break;
        }
    }

  argc -= optind;
  argv += optind;

  if (argc < 1)
    {
      show_help(progName);
      return EXIT_FAILURE;
    }

  const char *infile = argv[0];
  const char *outfile = argv[1];

  if (testMode)
    {
      /* Test mode: process the first image found in "indir" and
       * compare the GPU result to the CPU result.
       */
      std::string infilename(infile);
      std::string outfilename = outfile ? std::string(outfile) : std::string();

      run_test(infilename, outfilename);
    }
  else
    {
      /* Run experiment the requested number of times. */
      std::vector<std::string> files {infile};
      Experiment exp(files, 1, true);
      for (int i = 0; i < n_repeat; ++i)
        run_experiment(exp, std::string(infile),
                       outfile ? std::string(outfile) : std::string(),
                       silentMode);

      if (csvOutput)
        exp.outputCSVData(std::cout);
    }

  return EXIT_SUCCESS;
}
