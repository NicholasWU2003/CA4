#include "hip/hip_runtime.h"
/*
 * Skeleton code for use with Computer Architecture 2023 assignment 4,
 * LIACS, Leiden University.
 *
 * Task 2: tile composite. After solving task 2, your modified program
 * will be the template for tasks 3 and 4 (make copies!)
 *
 */

#include "image.h"
#include "experiment.hpp"

#include <iostream>

#include <cstring>
#include <cerrno>

#include <getopt.h>

#include <sys/types.h>
#include <dirent.h>

/* Some simple assert macro and inline function to handle CUDA-errors
 * more easily.
 */
#define CUDA_ASSERT(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void
cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
              file, line);

      if (abort)
        exit(code);
    }
}


/*
 * CUDA kernels
 *
 * "op_" functions below to be converted to CUDA kernels. Remember that
 * every kernel should be declared as `__global__ void`. Every function
 * you want to call from a kernel, should begin with `__device__`.
 *
 * op_cuda_copy is a CUDA kernel provided as example.
 */


/* Example kernel for an image copy operation. */
__global__ void
op_cuda_copy(uint32_t *dst, const uint32_t *src, const int rowstride,
             const int x, const int y,
             const int width, const int height)
{
  const int xx = blockIdx.x * blockDim.x + threadIdx.x;
  const int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx < x || xx >= width || yy < y || y >= height)
    return;

  /* Get the pixel in src and store in dst. */
  uint32_t pixel = *image_get_pixel_data(src, rowstride, xx, yy);
  *image_get_pixel_data(dst, rowstride, xx, yy) = pixel;
}


/*
 * CPU kernels
 *
 * Leave these functions in place, they are required for the "test
 * mode" to work. Make a copy in order to convert to a CUDA kernel.
 *
 */


/* Tiles the @tile image on @background using alpha blending. For the tile
 * an alpha value of @tile_alpha is used.
 */
/* Do not remove this function, it is required for the "test mode" to work. */
void
op_tile_composite(image_t *background,
                  const image_t *tile, const float tile_alpha)
{
  for (int y = 0; y < background->height; y++)
    {
      for (int x = 0; x < background->width; x++)
        {
          int tx = x % tile->width;
          int ty = y % tile->height;

          rgba_t dst, src;
          RGBA_unpack(dst, *image_get_pixel(background, x, y));
          RGBA_unpack(src, *image_get_pixel(tile, tx, ty));
          RGBA_mults(src, src, tile_alpha);
          RGBA_mults(dst, dst, 1.f - tile_alpha);
          RGBA_add(dst, dst, src);
          RGBA_pack(*image_get_pixel(background, x, y), dst);
        }
    }
}


__global__ void
op_tile_composite_noShared_batch(uint32_t *background, const uint32_t *tile,
                                 int bg_width, int bg_height,
                                 int tile_width, int tile_height,
                                 float tile_alpha, int nImages)
{
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    int total_height = bg_height * nImages;

    if (global_x >= bg_width || global_y >= total_height)
        return;

    int image_idx = global_y / bg_height;
    int local_y = global_y % bg_height;

    // Compute index into background array
    int pixel_idx = image_idx * (bg_width * bg_height) + local_y * bg_width + global_x;

    // Tile coordinates
    int tx = global_x % tile_width;
    int ty = (local_y) % tile_height;

    uint32_t bg_pix = background[pixel_idx];
    uint32_t tile_pix = tile[ty * tile_width + tx];

    rgba_t dst, src;
    RGBA_unpack(dst, bg_pix);
    RGBA_unpack(src, tile_pix);
    RGBA_mults(src, src, tile_alpha);
    RGBA_mults(dst, dst, 1.0f - tile_alpha);
    RGBA_add(dst, dst, src);
    RGBA_pack(background[pixel_idx], dst);
}


/* GPU kernel using shared memory to store the tile.
 * Here we assume tile_width = tile_height = 64.
 * Make sure your block size is (64,64).
 */
__global__ void
op_tile_composite_shared_batch(uint32_t *background, const uint32_t *tile,
                               int bg_width, int bg_height,
                               int tile_width, int tile_height,
                               float tile_alpha, int nImages)
{
    __shared__ uint32_t shared_tile[64][64];

    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    int total_height = bg_height * nImages;

    // Load tile into shared memory as before
    int local_idx = threadIdx.y * blockDim.x + threadIdx.x;
    int total_pixels = tile_width * tile_height; // e.g., 64*64=4096
    int stride = blockDim.x * blockDim.y;        // 32*32=1024 if that's your block size

    for (int i = local_idx; i < total_pixels; i += stride) {
        int ty = i / tile_width;
        int tx = i % tile_width;
        shared_tile[ty][tx] = tile[i];
    }

    __syncthreads();

    if (global_x >= bg_width || global_y >= total_height)
        return;

    int image_idx = global_y / bg_height;
    int local_y = global_y % bg_height;

    // Compute index into background array
    int pixel_idx = image_idx * (bg_width * bg_height) + local_y * bg_width + global_x;

    int tx = global_x % tile_width;
    int ty = local_y % tile_height;

    uint32_t background_px = background[pixel_idx];
    uint32_t tile_px = shared_tile[ty][tx];

    rgba_t dst, src;
    RGBA_unpack(dst, background_px);
    RGBA_unpack(src, tile_px);

    RGBA_mults(src, src, tile_alpha);
    RGBA_mults(dst, dst, 1.0f - tile_alpha);
    RGBA_add(dst, dst, src);

    RGBA_pack(background[pixel_idx], dst);
}






/* Returns elapsed time in msec */
static float
run_cuda_kernels(image_t *background[], const size_t nImages,
                 const image_t *tile)
{
  // Get dimensions
  size_t background_width = background[0]->width;
  size_t background_height = background[0]->height;

  // Calculate number of pixels in one image and total pixels in the batch
  size_t image_pixels = background_width * background_height;
  size_t total_pixels = image_pixels * nImages;

  // Byte sizes
  size_t image_bytes = image_pixels * sizeof(uint32_t);
  size_t total_bytes = total_pixels * sizeof(uint32_t);

  size_t tile_width = tile->width;
  size_t tile_height = tile->height;
  size_t tile_pixels = tile_width * tile_height;
  size_t tile_size = tile_pixels * sizeof(uint32_t);

  uint32_t *background_buffer;
  uint32_t *tile_buffer;

  // Allocate the combined buffer for all images in the batch
  CUDA_ASSERT(hipMalloc(&background_buffer, total_bytes));
  CUDA_ASSERT(hipMalloc(&tile_buffer, tile_size));

  // Copy all images into one large buffer on the device
  for (size_t i = 0; i < nImages; i++) {
    CUDA_ASSERT(hipMemcpy(background_buffer + i * image_pixels,
                           background[i]->data,
                           image_bytes,
                           hipMemcpyHostToDevice));
  }

  // Copy the tile
  CUDA_ASSERT(hipMemcpy(tile_buffer, tile->data, tile_size, hipMemcpyHostToDevice));

  int total_height = nImages * (int)background_height;
  dim3 block_size(32, 32);
  dim3 grid_size((background_width + block_size.x - 1) / block_size.x,
                 (total_height + block_size.y - 1) / block_size.y);

  hipEvent_t start, stop;
  CUDA_ASSERT(hipEventCreate(&start));
  CUDA_ASSERT(hipEventCreate(&stop));
  CUDA_ASSERT(hipEventRecord(start));

  // Launch batched kernel (using shared or no-shared variant)
  op_tile_composite_shared_batch<<<grid_size, block_size>>>(
    background_buffer,
    tile_buffer,
    (int)background_width,
    (int)background_height,
    (int)tile_width,
    (int)tile_height,
    0.2f,
    (int)nImages
  );

  // // Launch batched kernel (using shared or no-shared variant)
  // op_tile_composite_noShared_batch<<<grid_size, block_size>>>(
  //   background_buffer,
  //   tile_buffer,
  //   (int)background_width,
  //   (int)background_height,
  //   (int)tile_width,
  //   (int)tile_height,
  //   0.2f,
  //   (int)nImages
  // );

  CUDA_ASSERT(hipGetLastError());
  CUDA_ASSERT(hipEventRecord(stop));
  CUDA_ASSERT(hipEventSynchronize(stop));

  float msec = 0;
  CUDA_ASSERT(hipEventElapsedTime(&msec, start, stop));

  // Copy results back to host
  for (size_t i = 0; i < nImages; i++) {
    CUDA_ASSERT(hipMemcpy(background[i]->data,
                           background_buffer + i * image_pixels,
                           image_bytes,
                           hipMemcpyDeviceToHost));
  }

  // Free GPU memory
  CUDA_ASSERT(hipFree(background_buffer));
  CUDA_ASSERT(hipFree(tile_buffer));

  return msec;
}



static void
run_test(const std::string &infilename, image_t *tile,
         const std::string &outfilename)
{
  std::cout << "Testing with " << infilename << " ...\n";

  image_t *background[1];
  background[0] = image_new_from_pngfile(infilename.c_str());
  if (!background[0])
    return;

  /* Create a copy to be manipulated on CPU */
  image_t *original = image_new_from_image(background[0]);
  std::memcpy(original->data, background[0]->data,
              background[0]->rowstride * background[0]->height);

  /* Run CPU kernels */
  op_tile_composite(original, tile, 0.2f);

  /* Run GPU kernels */
  run_cuda_kernels(background, 1, tile);

  /* Compare the results */
  const int max_error = 64;
  int errors = image_compare(background[0], original, max_error);
  if (errors > 0)
    std::cerr << "Images do not match, " << errors
              << " errors detected (max=" << max_error << ").\n";
  else
    std::cerr << "Images match.\n";

  /* Write GPU result to PNG if requested */
  if (not outfilename.empty())
    {
      image_save_as_pngfile(background[0], outfilename.c_str());
      std::cerr << "Wrote GPU result to " << outfilename << "\n";
    }

  image_free(original);
  image_free(background[0]);
}


/* Process a single image, or set/batch of images.. Warning: does not detect
 * errors.
 */
static void
process_images(size_t i, ExperimentTimer &timer,
               const std::vector<std::string> &infilenames,
               const image_t *tile,
               const std::vector<std::string> &outfilenames)
{
  /* Load image */
  image_t *background[infilenames.size()];

  auto startTime = ExperimentTimer::now();
  for (size_t j = 0; j < infilenames.size(); ++j)
    {
      background[j] = image_new_from_pngfile(infilenames[j].c_str());
      if (!background[j])
        return;
    }
  auto endTime = ExperimentTimer::now();

  timer.setLoadTime(i, endTime, startTime);

  float msec = run_cuda_kernels(background, infilenames.size(), tile);
  timer.setComputeTime(i, msec / 1000.);

  /* Save results if desired and if applicable */
  if (not outfilenames.empty())
    {
      for (size_t j = 0; j < outfilenames.size(); ++j)
        image_save_as_pngfile(background[j], outfilenames[j].c_str());
    }

  for (size_t j = 0; j < infilenames.size(); ++j)
    image_free(background[j]);
}


/* Code to run a single experiment, depending on batchSize parameter. */
static void
run_experiment(Experiment &exp,
               const std::string &indir, const std::string &outdir,
               image_t *tile, bool silentMode)
{
  ExperimentTimer timer(exp.addMeasurement());

  timer.start();

  for (size_t i = 0; i < exp.getNBatches(); ++i)
    {
      size_t count = exp.getBatchSize(i);

      std::vector<std::string> infilenames;
      std::vector<std::string> outfilenames;

      for (size_t j = 0; j < count; ++j)
        {
          infilenames.emplace_back(indir + std::string("/") + exp.getFrameFile(i, j));
          if (not outdir.empty())
            outfilenames.emplace_back(outdir + std::string("/") + exp.getFrameFile(i, j));
        }

      if (not silentMode)
        {
          if (count == 1)
            std::cout << "Processing " << infilenames[0] << " ...\n" << std::flush;
          else
            std::cout << "Processing " << infilenames[0] << " - "
                      << infilenames[count - 1] << "...\n" << std::flush;
        }

      process_images(i, timer, infilenames, tile, outfilenames);
    }

  /* Note that the full timing of the experiment will include image
   * loading & saving time and memory transfers to and from the GPU.
   * The memory transfers are not counted in the runtime of the
   * individual images.
   */
  timer.end();

  /* Print statistics */
  if (not silentMode)
    {
      std::cout << "====\n";
      exp.printCPUUtilization(exp.back(), std::cout);
      exp.printStatistics(exp.back(), std::cout);
      std::cout << "====\n";
    }
}


/*
 * Main function
 */

static bool
has_png_extension(const std::string &filename)
{
  if (filename.size() < 4)
    return false;

  return filename.substr(filename.size() - 4, 4) == ".png";
}

static void
show_help(const char *progName)
{
  std::cerr << "usage: " << progName << " [-t] [-s] [-c] [-r REPEAT] [-b BATCH_SIZE] [-1] <indir> <tilefile> [outdir]\n"
            << "\n  where <indir>, and [outdir] are directories containing PNG files."
            << "\n  <tilefile> is a PNG file."
            << "\n  [outdir] is an optional parameter.\n"
            << "\n  -t  test mode: only processes the first image found in <indir> and compares"
            << "\n      the GPU result to the CPU result. Optionally outputs the GPU result if"
            << "\n      outdir is specified."
            << "\n  -s  disables output of experiment summaries (silent mode)."
            << "\n  -c  outputs an overview of all experiment results in CSV format."
            << "\n  -r  configures the number of times the experiment is repeated."
            << "\n  -b  configures the batch size (defaults to 1)."
            << "\n  -1  stops the experiment after processing one batch (defaults to false).\n";
}


int
main(int argc, char **argv)
{
  char c;
  long int n_repeat = 1;
  long int batchSize = 1;
  bool singleBatch = false;
  bool csvOutput = false;
  bool silentMode = false;
  bool testMode = false;

  /* Command line parsing */
  const char *progName = argv[0];

  while ((c = getopt(argc, argv, "r:b:1cst")) != -1)
    {
      switch (c)
        {
          case 'r':
            n_repeat = std::strtol(optarg, NULL, 10);
            if (errno > 0 || n_repeat == 0)
              {
                std::cerr << "Could not convert n_repeat argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case 'b':
            batchSize = std::strtol(optarg, NULL, 10);
            if (errno > 0 || batchSize == 0)
              {
                std::cerr << "Could not convert batchSize argument to integer.\n";
                exit(EXIT_FAILURE);
              }
            break;

          case '1':
            singleBatch = true;
            break;

          case 'c':
            csvOutput = true;
            break;

          case 's':
            silentMode = true;
            break;

          case 't':
            testMode = true;
            break;

          case 'h':
          default:
            show_help(progName);
            return EXIT_FAILURE;
            break;
        }
    }

  argc -= optind;
  argv += optind;

  if (argc < 2)
    {
      show_help(progName);
      return EXIT_FAILURE;
    }

  const char *indir = argv[0];
  const char *tilefile = argv[1];
  const char *outdir = argv[2];

  /* Check and open directories */
  DIR *indirp = opendir(indir);
  if (indirp == NULL)
    {
      const char *err = strerror(errno);
      std::cerr << "error: could not open directory '" << indir
                << "': " << err << "\n";
      return EXIT_FAILURE;
    }

  DIR *outdirp = NULL;
  if (outdir)
    {
      outdirp = opendir(outdir);
      if (outdirp == NULL)
        {
          const char *err = strerror(errno);
          std::cerr << "error: could not open directory '" << outdir
              << "': " << err << "\n";
          closedir(indirp);
          return EXIT_FAILURE;
        }

      closedir(outdirp);
    }

  /* Createa a list pf PNG files in given directory. */
  std::vector<std::string> framefiles;
  for (struct dirent *ent = readdir(indirp);
       ent != NULL; ent = readdir(indirp))
    {
      std::string filename(ent->d_name);
      if (has_png_extension(filename))
        framefiles.emplace_back(std::move(filename));
    }

  closedir(indirp);

  /* Load tile image */
  image_t *tile = image_new_from_pngfile(tilefile);
  if (!tile)
    return EXIT_FAILURE;

  if (testMode)
    {
      /* Test mode: process the first image found in "indir" and
       * compare the GPU result to the CPU result.
       */
      std::string infilename = std::string(indir) + std::string("/") + framefiles[0];
      std::string outfilename = outdir ? std::string(outdir) + std::string("/") + framefiles[0] : std::string();

      run_test(infilename, tile, outfilename);
    }
  else
    {
      /* Run experiment the requested number of times. */
      Experiment exp(framefiles, batchSize, singleBatch);
      for (int i = 0; i < n_repeat; ++i)
        run_experiment(exp, std::string(indir),
                       outdir ? std::string(outdir) : std::string(),
                       tile, silentMode);

      if (csvOutput)
        exp.outputCSVData(std::cout);
    }

  /* Clean up */
  image_free(tile);

  return EXIT_SUCCESS;
}
